
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void MatrixMul( int*M, int*N, int*P, int Width)
{
	int tid, tx, ty;
	tx = blockDim.x*blockIdx.x + threadIdx.x;
	ty = blockDim.y*blockIdx.y + threadIdx.y;
	tid = Width*ty + tx;

	int Value = 0;
	int MVal = 0;
	int NVal = 0;

	for (int i = 0; i < Width; i++)
	{
		MVal = M[ty * Width + i];
		NVal = N[i * Width + tx];
		Value += MVal * NVal;
	}

	P[tid] = Value;
}

extern "C"
int* kernel(int *M, int *N,int BufferSize)
{
	int *P_cuda;

	P_cuda = (int*)malloc(BufferSize);
	//P_cuda = new int[BufferSize/sizeof(int)];
	memset((void*)P_cuda, 0, BufferSize);
	int *dev_M;
	int *dev_N;	
	int *dev_P;	
	
	hipMalloc((void**)&dev_M, BufferSize);
	hipMalloc((void**)&dev_N, BufferSize);
	hipMalloc((void**)&dev_P, BufferSize);

	hipMemcpy(dev_M, M, BufferSize, hipMemcpyHostToDevice);
	hipMemcpy(dev_N, N, BufferSize, hipMemcpyHostToDevice);

	dim3 Dg(3, 4, 1);
	dim3 Db(4, 3, 1);
    MatrixMul<<<Dg,Db>>>(dev_M, dev_N, dev_P, 12);
	
	hipMemcpy(P_cuda, dev_P, BufferSize, hipMemcpyDeviceToHost);

	hipFree(dev_M);
	hipFree(dev_N);
	hipFree(dev_P);

	return P_cuda;
}

